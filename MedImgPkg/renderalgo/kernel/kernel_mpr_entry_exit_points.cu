#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <hip/device_functions.h>

#include "arithmetic/mi_cuda_graphic.h"
#include "renderalgo/mi_ray_caster_define.h"

inline __device__  float ray_intersect_brick(float3 init_pt, float3 brick_min, float3 brick_dim, float3 ray_dir, float* start_step, float* end_step) {
    float3 ray_r = 1.0f/ray_dir;
    float3 bottom = (brick_min - init_pt);
    float3 top = (brick_min + brick_dim - init_pt);
    float3 tbot = ray_r * bottom;
    float3 ttop = ray_r * top;

    //Adjust
    if (fabs(bottom.x) < INF) {
        tbot.x = 0.0;
    }
    if (fabs(bottom.y) < INF) {
        tbot.y = 0.0;
    }
    if (fabs(bottom.z) < INF) {
        tbot.z = 0.0;
    }

    if (fabs(top.x) < INF) {
        ttop.x = 0.0;
    }
    if (fabs(top.y) < INF) {
        ttop.y = 0.0;
    }
    if (fabs(top.z) < INF) {
        ttop.z = 0.0;
    }

    float3 tmin = fminf(tbot, ttop);
    float3 tmax = fmaxf(tbot, ttop);
    float tnear = fmaxf(fmaxf(tmin.x, tmin.y), tmin.z);
    float tfar = fminf(fminf(tmax.x, tmax.y), tmax.z);

    *start_step = tnear;
    *end_step = tfar;

    return tnear - *start_step;
}

inline __device__ bool outside(float3 pt , float3 bound) {
    if (pt.x < -INF || pt.y < -INF || pt.z < -INF) {
        return true;
    }
    if (pt.x > bound.x + INF || pt.y > bound.y + INF || pt.z > bound.z + INF) {
        return true;
    }
    return false;
}

__global__ void kernel_calculate_mpr_entry_exit_points(hipSurfaceObject_t entry_surf, hipSurfaceObject_t exit_surf, 
    int width, int height, mat4 mat_mvp_inv,  float3 volume_dim, float thickness, float3 ray_dir ) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x > width - 1 || y > height - 1) {
        return;
    }

    float ndc_x = (float(x) + 0.5f) / float(width);
    float ndc_y = (float(y) + 0.5f) / float(height);
    float3 pos_ndc = make_float3(ndc_x * 2.0f - 1.0f, ndc_y * 2.0f - 1.0f, 1.0f);
    float4 central4 = mat_mvp_inv * make_float4(pos_ndc, 1.0f);
    float3 central = make_float3(central4/ central4.w);

    float3 entry_point, exit_point;
    if (thickness <= 1.0f) {
        entry_point = central;
        exit_point = central + ray_dir * thickness;
    } else {
        entry_point = central - ray_dir * thickness * 0.5f;
        exit_point = central + ray_dir * thickness * 0.5f;
    }

    float entry_step = 0.0f;
    float exit_step = 0.0f;
    float3 entry_intersection = entry_point;
    float3 exit_intersection = exit_point;
    ray_intersect_brick(entry_point, make_float3(0.0f, 0.0f, 0.0f), volume_dim, ray_dir, &entry_step, &exit_step);

    //Entry point outside
    if (outside(entry_point, volume_dim - make_float3(1.0f))) {
        if (entry_step >= exit_step || entry_step < 0 || entry_step > thickness)// check entry points in range of thickness and volume
        {
            exit_step = -1.0f;
            surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, -1.0f), entry_surf, x * 4, y);
            surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, -1.0f), exit_surf, x * 4, y);
            return;
        }
        entry_intersection = entry_point + entry_step * ray_dir;
    }

    //Exit point outside
    if (outside(exit_point, volume_dim - make_float3(1.0f)))
    {
        if (entry_step >= exit_step)
        {
            exit_step = -1.0f;
            surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, -1.0f), entry_surf, x * 4, y);
            surf2Dwrite(make_float4(0.0f, 0.0f, 0.0f, -1.0f), exit_surf, x * 4, y);
            return;
        }
        exit_intersection = entry_point + exit_step * ray_dir;
        if (thickness <= 1.0f)//forbid border exit_step=0 : entry == exit (ray direction is Nan)
        {
            exit_intersection = entry_point + thickness * ray_dir;
        }
    }

    surf2Dwrite(make_float4(entry_intersection, -1.0f), entry_surf, x * 4, y);
    surf2Dwrite(make_float4(exit_intersection, -1.0f), exit_surf, x * 4, y);
}

extern "C" 
hipError_t calculate_mpr_entry_exit_points(hipSurfaceObject_t entry_surf, hipSurfaceObject_t exit_surf,
    int width, int height, mat4 mat_mvp_inv, float3 volume_dim, float thickness, float3 ray_dir) {
    
    const int BLOCK_DIM = 16;
    dim3 block(BLOCK_DIM, BLOCK_DIM);
    dim3 grid(width / BLOCK_DIM, height / BLOCK_DIM);
    if (grid.x * BLOCK_DIM != width) {
        grid.x += 1;
    }
    if (grid.y * BLOCK_DIM != height) {
        grid.y += 1;
    }

    kernel_calculate_mpr_entry_exit_points <<<grid,block>>>(entry_surf, exit_surf, 
        width, height, mat_mvp_inv, volume_dim, thickness, ray_dir);

    return hipDeviceSynchronize();
}