#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "arithmetic/mi_cuda_math.h"

__global__ void kernel_surface_2d_rgba8_flip_vertical_to_global_memory_rgb8(hipSurfaceObject_t sur_rgba8, int width, int height, unsigned char* d_rgb_8) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < width && y < height) {
        uchar4 rgba;
        surf2Dread(&rgba, sur_rgba8, x * 4, y);
        unsigned int idx = (height - y) * width + x;
        d_rgb_8[idx * 3] = rgba.x;
        d_rgb_8[idx * 3 + 1] = rgba.y;
        d_rgb_8[idx * 3 + 2] = rgba.z;
    }
}

extern "C"
hipError_t surface_2d_rgba8_flip_vertical_to_global_memory_rgb8(hipSurfaceObject_t sur_rgba8, int width, int height, unsigned char* d_rgb_8) {
    const int BLOCKDIM = 16;
    dim3 block_dim(BLOCKDIM, BLOCKDIM);
    dim3 grid_dim(width / BLOCKDIM, height / BLOCKDIM);

    kernel_surface_2d_rgba8_flip_vertical_to_global_memory_rgb8 << <grid_dim, block_dim >> >(sur_rgba8, width, height, d_rgb_8);

    return hipDeviceSynchronize();
}