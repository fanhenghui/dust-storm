#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "arithmetic/mi_cuda_math.h"
#include "renderalgo/mi_brick_define.h"

using namespace medical_imaging;


__device__ void kernel_statistic_volume_info_in_cube(int3 begin, int3 end, hipTextureObject_t volume_tex, float *min0, float* max0) {
    unsigned short min00 = 65535;
    unsigned short max00 = 0;
    unsigned short v = 0;
    for (int z = begin.z; z < end.z; ++z) {
        for (int y = begin.y; y < end.y; ++y) {
            for (int x = begin.x; x < end.x; ++x) {
                v = tex3D<unsigned short>(volume_tex, x, y, z);
                min00 = min(min00, v);
                max00 = max(max00, v);
            }
        }
    }
    *min0 = float(min00);
    *max0 = float(max00);
}

__global__ void kernel_cuda_calculate_volume_brick_info(hipTextureObject_t volume_tex, dim3 volume_dim,
    int brick_size, dim3 brick_dim, int brick_margin, VolumeBrickInfo* d_data) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x > brick_dim.x - 1 || y > brick_dim.y - 1 || z > brick_dim.z - 1) {
        return;
    }

    unsigned int idx = z * brick_dim.x * brick_dim.y + y * brick_dim.x + x;
    int3 begin = make_int3(x*brick_size , y*brick_size, z*brick_size);
    int3 end = begin + make_int3(brick_size);
    int3 margin = make_int3(brick_margin);

    begin -= margin;
    end += margin;

    begin = max(begin, make_int3(0));
    begin = min(begin, make_int3(volume_dim));
    
    end = max(end, make_int3(0));
    end = min(end, make_int3(volume_dim));

    float max0 = 0.0f;
    float min0 = 0.0f;
    kernel_statistic_volume_info_in_cube(begin, end, volume_tex, &min0, &max0);
    d_data[idx].min = min0;
    d_data[idx].max = max0;
}

__shared__ unsigned char _s_visible_labels[256];
__device__ void kernel_statistic_mask_info_in_cube(int3 begin, int3 end, hipTextureObject_t mask_tex, int* label_code) {
    unsigned char label_max = 0;
    unsigned char label_min = 255;
    unsigned char label = 0;
    int all_air = 1;
    for (int z = begin.z; z < end.z; ++z) {
        for (int y = begin.y; y < end.y; ++y) {
            for (int x = begin.x; x < end.x; ++x) {
                label = tex3D<unsigned char>(mask_tex, x, y, z);
                if (label == 0) {
                    continue;
                }
                if (_s_visible_labels[label] != 0) {
                    label_min = min(label_min, label);
                    label_max = max(label_max, label);
                    all_air ^= all_air;
                }
                
                
            }
        }
    }

    if (1 == all_air) {
        *label_code = 0;
    } else if (label_min == label_max) {
        *label_code = (int)label_min;
    } else {
        *label_code = 255;
    }
}

__global__ void kernel_cuda_calculate_mask_brick_info(hipTextureObject_t mask_tex, dim3 mask_dim,
    int brick_size, dim3 brick_dim, int brick_margin,
    dim3 brick_range_min, dim3 brick_range_dim,
    unsigned char* d_visible_label_bucket, int visible_label_bucket_count,
    MaskBrickInfo* d_data) {
    //-------------------------------------------------------------//
    //set shared memory
    if (threadIdx.y * blockDim.x * blockDim.y + threadIdx.x == 0) {
        //reset shared data to 0
        int* shared_map = (int*)_s_visible_labels;
        for (int i = 0; i < 64; ++i) {
            shared_map[i] ^= shared_map[i];
        }
        //write global to shared visible labels
        for (int i = 0; i < visible_label_bucket_count; ++i) {
            _s_visible_labels[i] = d_visible_label_bucket[i];
        }
    }

    __syncthreads();
    //-------------------------------------------------------------//

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int z = blockIdx.z * blockDim.z + threadIdx.z;
    if (x > brick_range_dim.x - 1 || y > brick_range_dim.y - 1 || z > brick_range_dim.z - 1) {
        return;
    }

    x += brick_range_min.x;
    y += brick_range_min.y;
    z += brick_range_min.z;
    unsigned int idx = z * brick_dim.x * brick_dim.y + y * brick_dim.x + x;
    
    int3 begin = make_int3(x*brick_size, y*brick_size, z*brick_size);
    int3 end = begin + make_int3(brick_size);
    int3 margin = make_int3(brick_margin);

    begin -= margin;
    end += margin;

    begin = max(begin, make_int3(0));
    begin = min(begin, make_int3(mask_dim));

    end = max(end, make_int3(0));
    end = min(end, make_int3(mask_dim));

    int label_code = 0;
    kernel_statistic_mask_info_in_cube(begin, end, mask_tex, &label_code);
    d_data[idx].label = label_code;
}

extern "C"
hipError_t cuda_calculate_volume_brick_info(hipTextureObject_t volume_tex, dim3 volume_dim,
    int brick_size, dim3 brick_dim, int brick_margin, VolumeBrickInfo* d_data) {
    dim3 block_dim(5, 5, 1);
    dim3 grid_dim(
        brick_dim.x / block_dim.x,
        brick_dim.y / block_dim.y,
        brick_dim.z / block_dim.z);
    if (grid_dim.x * block_dim.x != brick_dim.x) {
        grid_dim.x += 1;
    }
    if (grid_dim.y * block_dim.y != brick_dim.y) {
        grid_dim.y += 1;
    }
    if (grid_dim.z * block_dim.z != brick_dim.z) {
        grid_dim.z += 1;
    }
    
    kernel_cuda_calculate_volume_brick_info << <grid_dim, block_dim >> > (volume_tex, volume_dim,
        brick_size, brick_dim, brick_margin, d_data);

    return hipDeviceSynchronize();
}

extern "C"
hipError_t cuda_calculate_mask_brick_info(hipTextureObject_t mask_tex, dim3 mask_dim,
    int brick_size, dim3 brick_dim, int brick_margin, 
    dim3 brick_range_min, dim3 brick_range_dim,
    unsigned char* d_visible_label_bucket, int visible_label_bucket_count,
    MaskBrickInfo* d_data) {
    
    dim3 block_dim(8, 8, 1);
    dim3 grid_dim(
        brick_dim.x / block_dim.x,
        brick_dim.y / block_dim.y,
        brick_dim.z / block_dim.z);
    if (grid_dim.x * block_dim.x != brick_dim.x) {
        grid_dim.x += 1;
    }
    if (grid_dim.y * block_dim.y != brick_dim.y) {
        grid_dim.y += 1;
    }
    if (grid_dim.z * block_dim.z != brick_dim.z) {
        grid_dim.z += 1;
    }

    kernel_cuda_calculate_mask_brick_info << <grid_dim, block_dim >> > (mask_tex, mask_dim, 
        brick_size, brick_dim, brick_margin, 
        brick_range_min, brick_range_dim, 
        d_visible_label_bucket, visible_label_bucket_count, d_data);

    return hipDeviceSynchronize();
}