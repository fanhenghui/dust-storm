#include "hip/hip_runtime.h"
#include "GL/glew.h"

//CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>  
#include <cuda_gl_interop.h>
#include <hip/hip_texture_types.h>
#include <hip/hip_vector_types.h>
#include <math_functions.h>
#include <vector_functions.h>

#include <iostream>
#include <fstream>
#include <sstream>

#include "mi_cuda_vr.h"
#include "arithmetic/mi_cuda_math.h"

//-------------------------------------------//
//Global Parameter Define

#define CHECK_CUDA_ERROR {\
hipError_t err = hipGetLastError(); \
if (err != hipSuccess) {\
    std::cout << "CUDA error: " << err << " in function: " << __FUNCTION__ <<\
    " line: " << __LINE__ << std::endl; \
}}\

//-------------------------------------------//


__device__ void composite() { 

}

__device__ void kernel_ray_cast(float3 ray_dir, float start_step, float end_step, cudaVolumeInfos volume_infos, float4* color_norm) {
    float3 sample_pos;
    for (float i = start_step; i < end_step; i+=1.0) {
        sample_pos = start_step + ray_dir*i;
        //composite();
        if ((*color_norm).w > 0.95) {
            (*color_norm).w = 1.0;
            break;
        }
    }
}

__device__ int kernel_preprocess(float3 entry, float3 exit, float sample_step, float3* ray_start, float3* ray_dir, float* start_step, float* end_step) {
    float3 ray_dir0 = entry - exit;
    float3 ray_dir_norm = normalize(ray_dir0);
    float ray_length = length(ray_dir0);
    if(ray_length < 1e-5) {
        return -1;
    } 

    *ray_start = entry;
    *ray_dir = ray_dir0*make_float3(sample_step);
    *start_step = 0;
    *end_step = ray_length/sample_step;

    return 0;
}

__global__ void kernel_ray_cast_main(cudaGLTexture entry_tex, cudaGLTexture exit_tex, cudaVolumeInfos volume_info, cudaRayCastInfos ray_cast_info, unsigned char* result) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > entry_tex.width-1 || y > entry_tex.height-1) {
        return;
    }
    unsigned int idx = y*entry_tex.width + x;

    float4 entry = tex2D<float4>(entry_tex.cuda_tex_obj, x, y);
    float4 exit  = tex2D<float4>(exit_tex.cuda_tex_obj, x, y);

    float3 entry3 = make_float3(entry);
    float3 exit3 = make_float3(exit);

    float3 ray_start, ray_dir;
    float start_step, end_step;

    if(0 != kernel_preprocess(entry3, exit3, ray_cast_info.sample_step, &ray_start, &ray_dir, &start_step, &end_step)) {
        result[idx*4] = 0;
        result[idx*4+1] = 0;
        result[idx*4+2] = 0;
        result[idx*4] = 0;
        return;
    }
    
    result[idx*4] = entry.x/volume_info.dim.x*255;
    result[idx*4+1] = entry.y/volume_info.dim.y*255;
    result[idx*4+2] = entry.z/volume_info.dim.z*255;
    result[idx*4+3] = 255;
}

//result will be one of color, JEPG buffer.
extern "C"  
int ray_cast(cudaGLTexture entry_tex, cudaGLTexture exit_tex, cudaVolumeInfos volume_info, cudaRayCastInfos ray_cast_info, unsigned char* d_result, unsigned char* h_result) {
    //1 launch ray cast kernel
    int width = entry_tex.width;
    int height = entry_tex.height;

    hipGraphicsMapResources(1, &entry_tex.cuda_res);
    hipGraphicsMapResources(1, &exit_tex.cuda_res);

    CHECK_CUDA_ERROR; 
    #define BLOCK_SIZE 16
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(width / BLOCK_SIZE, height / BLOCK_SIZE);
    kernel_ray_cast_main<<<grid, block>>>(entry_tex, exit_tex, volume_info, ray_cast_info, d_result);

    //2 JPEG compress(optional)


    //3 Memcpy device result to host, and return
    hipDeviceSynchronize();
    hipGraphicsUnmapResources(1, &entry_tex.cuda_res);
    hipGraphicsUnmapResources(1, &exit_tex.cuda_res);

    hipMemcpy(h_result, d_result, width*height*4, hipMemcpyDeviceToHost);
    CHECK_CUDA_ERROR; 

    return 0;
}

extern "C"
int bind_gl_texture(cudaGLTexture& gl_cuda_tex) {
    //1 register GL texture to CUDA Graphic resource
    gl_cuda_tex.cuda_res = NULL;
    hipGraphicsGLRegisterImage(&gl_cuda_tex.cuda_res, gl_cuda_tex.gl_tex_id, gl_cuda_tex.target, hipGraphicsRegisterFlagsReadOnly);

    CHECK_CUDA_ERROR;

    //2 map the graphic resource to CUDA array
    gl_cuda_tex.d_cuda_array = NULL;
    hipGraphicsMapResources(1, &gl_cuda_tex.cuda_res);
    hipGraphicsSubResourceGetMappedArray(&gl_cuda_tex.d_cuda_array, gl_cuda_tex.cuda_res, 0,0);
    CHECK_CUDA_ERROR;

    //3 create CUDA texture by CUDA array
    //CUDA resource
    struct hipResourceDesc res_desc;
    memset(&res_desc, 0, sizeof(hipResourceDesc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = gl_cuda_tex.d_cuda_array;

    struct hipTextureDesc tex_desc;
    memset(&tex_desc, 0, sizeof(hipTextureDesc));
    tex_desc.addressMode[0] = hipAddressModeWrap;
    tex_desc.addressMode[1] = hipAddressModeWrap;
    tex_desc.filterMode = hipFilterModePoint;
    tex_desc.readMode = hipReadModeElementType;
    tex_desc.normalizedCoords = 0;
    
    hipCreateTextureObject(&gl_cuda_tex.cuda_tex_obj, &res_desc, &tex_desc, NULL);

    CHECK_CUDA_ERROR;

    hipGraphicsUnmapResources(1, &gl_cuda_tex.cuda_res);

    CHECK_CUDA_ERROR;

    return 0;
}

extern "C"
int init_data(cudaVolumeInfos& cuda_volume_infos, unsigned short* data, unsigned int* dim) {
    const unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(unsigned short);

    cuda_volume_infos.dim[0] = dim[0];
    cuda_volume_infos.dim[1] = dim[1];
    cuda_volume_infos.dim[2] = dim[2];

    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(16,0,0,0,hipChannelFormatKindUnsigned);
    CHECK_CUDA_ERROR;

    hipExtent extent;
    extent.width = dim[0];
    extent.height = dim[1];
    extent.depth = dim[2];
    hipMalloc3DArray(&cuda_volume_infos.d_volume_array, &channel_desc, extent);

    CHECK_CUDA_ERROR;

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void *)data,extent.width*sizeof(unsigned short), extent.width, extent.height);
    copyParams.dstArray = cuda_volume_infos.d_volume_array;
    copyParams.extent   = extent;
    copyParams.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    CHECK_CUDA_ERROR;

    //Cuda resource
    struct hipResourceDesc  res_desc;
    memset(&res_desc, 0, sizeof(hipResourceDesc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = cuda_volume_infos.d_volume_array;
    
    //Texture parameter (like GL's glTexParameteri)
    struct hipTextureDesc tex_desc;
    memset(&tex_desc,0, sizeof(hipTextureDesc));
    tex_desc.addressMode[0] = hipAddressModeWrap;
    tex_desc.addressMode[1] = hipAddressModeWrap;
    tex_desc.addressMode[2] = hipAddressModeWrap;
    tex_desc.filterMode = hipFilterModePoint;
    tex_desc.readMode = hipReadModeNormalizedFloat;
    tex_desc.normalizedCoords = 1;

    //create texture
    hipTextureObject_t tex_obj = 0;
    hipCreateTextureObject(&cuda_volume_infos.volume_tex_obj, &res_desc, &tex_desc, NULL);

    CHECK_CUDA_ERROR;

    return 0;
}



