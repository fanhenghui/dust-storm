#include "hip/hip_runtime.h"
#include "GL/glew.h"

//CUDA includes
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>  
#include <cuda_gl_interop.h>
#include <hip/hip_texture_types.h>
#include <hip/hip_vector_types.h>
#include <math_functions.h>
#include <vector_functions.h>

#include <iostream>
#include <fstream>
#include <sstream>

#include "mi_cuda_graphic.h"
#include "arithmetic/mi_cuda_math.h"

__device__ float4 kernel_ray_cast(cudaVolumeInfos* volume_infos, cudaRayCastInfos* ray_cast_infos, float3 ray_dir, float3 ray_start, float start_step, float end_step, float4 input_color) {
    float4 integral_color = input_color;
    float3 sample_pos;
    float3 sample_norm;
    float4 color_ori;
    const float3 dim3_r = make_float3(1.0/volume_infos->dim.x, 1.0/volume_infos->dim.y, 1.0/volume_infos->dim.z);
    float ww,wl,min_gray,gray;
    for (float i = start_step; i < end_step; i+=1.0) {
        sample_pos = ray_start + ray_dir*i;
        sample_norm = sample_pos*dim3_r;

        ww = ray_cast_infos->d_wl_array[0];
        wl = ray_cast_infos->d_wl_array[1];

        min_gray = wl - ww*0.5;

        ///Composite
        gray= tex3D<float>(volume_infos->volume_tex_obj, sample_norm.x,sample_norm.y,sample_norm.z);
        gray = (gray - min_gray)/ww;
        gray = clamp(gray,0.0,1.0);
        color_ori = tex1D<float4>(ray_cast_infos->lut_tex_obj, gray);
        if (color_ori.w > 0.0) {
            integral_color.x += color_ori.x * color_ori.w*(1-integral_color.w);
            integral_color.y += color_ori.y * color_ori.w*(1-integral_color.w);
            integral_color.z += color_ori.z * color_ori.w*(1-integral_color.w);
            integral_color.w += color_ori.w *(1-integral_color.w);
        }

        if (integral_color.w > 0.95) {
            integral_color.w = 1.0;
            break;
        }
    }

    return integral_color;
}

__device__ int kernel_preprocess(float3 entry, float3 exit, float sample_step, float3* ray_start, float3* ray_dir, float* start_step, float* end_step) {
    float3 ray_dir0 = exit - entry;
    float3 ray_dir_norm = normalize(ray_dir0);
    float ray_length = length(ray_dir0);
    if(ray_length < 1e-5) {
        return -1;
    } 

    *ray_start = entry;
    *ray_dir = ray_dir_norm*make_float3(sample_step);
    *start_step = 0;
    *end_step = ray_length/sample_step;

    return 0;
}

__global__ void kernel_ray_cast_main(hipTextureObject_t entry_tex, hipTextureObject_t exit_tex, int width, int height, cudaVolumeInfos volume_infos, cudaRayCastInfos ray_cast_infos, unsigned char* result) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > width-1 || y > height-1) {
        return;
    }
    uint idx = y*width + x;

    float4 entry = tex2D<float4>(entry_tex, x, y);
    float4 exit  = tex2D<float4>(exit_tex, x, y);

    float3 entry3 = make_float3(entry);
    float3 exit3 = make_float3(exit);

    float3 ray_start, ray_dir;
    float start_step, end_step;

    /////////////////////////////////////////
    //debug
    //result[idx*4] = exit.x/volume_infos.dim.x*255;
    //result[idx*4+1] = exit.y/volume_infos.dim.y*255;
    //result[idx*4+2] = exit.z/volume_infos.dim.z*255;
    //result[idx*4+3] = 255;

    //return;
    /////////////////////////////////////////

    if(0 != kernel_preprocess(entry3, exit3, ray_cast_infos.sample_step, &ray_start, &ray_dir, &start_step, &end_step)) {
        result[idx*4] = 0;
        result[idx*4+1] = 0;
        result[idx*4+2] = 0;
        result[idx*4] = 0;
        return;
    }

    //__syncthreads();

    float4 input_color = make_float4(0);
    float4 integral_color = kernel_ray_cast(&volume_infos, &ray_cast_infos, ray_dir, ray_start, start_step, end_step, input_color );
    
    //__syncthreads();
    clamp(integral_color,0.0,1.0);
    result[idx*4] = integral_color.x*255;
    result[idx*4+1] = integral_color.y*255;
    result[idx*4+2] = integral_color.z*255;
    result[idx*4+3] = 255;
}

__global__ void kernel_ray_cast_main_whole(hipTextureObject_t entry_tex, hipTextureObject_t exit_tex, int width, int height,  cudaVolumeInfos volume_infos, cudaRayCastInfos ray_cast_infos, unsigned char* result) {
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x > width-1 || y > height-1) {
        return;
    }
    uint idx = y*width + x;

    float4 entry = tex2D<float4>(entry_tex, x, y);
    float4 exit  = tex2D<float4>(exit_tex, x, y);

    float3 entry3 = make_float3(entry);
    float3 exit3 = make_float3(exit);

    float3 ray_start, ray_dir;
    float start_step, end_step;

    /////////////////////////////////////////
    //debug
    //result[idx*4] = exit.x/volume_infos.dim.x*255;
    //result[idx*4+1] = exit.y/volume_infos.dim.y*255;
    //result[idx*4+2] = exit.z/volume_infos.dim.z*255;
    //result[idx*4+3] = 255;

    //return;
    /////////////////////////////////////////

    float3 ray_dir0 = exit3 - entry3;
    float3 ray_dir_norm = normalize(ray_dir0);
    float ray_length = length(ray_dir0);
    if(ray_length < 1e-5) {
        result[idx*4] = 0;
        result[idx*4+1] = 0;
        result[idx*4+2] = 0;
        result[idx*4+3] = 0;
        return;
    } 

    ray_start = entry3;
    ray_dir = ray_dir_norm*make_float3(ray_cast_infos.sample_step);
    start_step = 0;
    end_step = ray_length/ray_cast_infos.sample_step;

    //__syncthreads();

    float ww = ray_cast_infos.d_wl_array[0];
    float wl = ray_cast_infos.d_wl_array[1];
    float min_gray = wl - ww*0.5;
    float3 dim3_r = make_float3(1.0/volume_infos.dim.x, 1.0/volume_infos.dim.y, 1.0/volume_infos.dim.z);

    float4 integral_color = make_float4(0);
    float3 sample_pos;
    float3 sample_norm;
    float4 color_ori;
    float gray;
    for (float i = start_step; i < end_step; i+=1.0) {
        sample_pos = ray_start + ray_dir*i;
        sample_norm = sample_pos*dim3_r;

        ///Composite
        gray= tex3D<float>(volume_infos.volume_tex_obj, sample_norm.x,sample_norm.y,sample_norm.z);
        gray = (gray - min_gray)/ww;
        gray = clamp(gray,0.0,1.0);
        color_ori = tex1D<float4>(ray_cast_infos.lut_tex_obj, gray);
        if (color_ori.w > 0.0) {
            integral_color.x += color_ori.x * color_ori.w*(1-integral_color.w);
            integral_color.y += color_ori.y * color_ori.w*(1-integral_color.w);
            integral_color.z += color_ori.z * color_ori.w*(1-integral_color.w);
            integral_color.w += color_ori.w *(1-integral_color.w);
        }


        if (integral_color.w > 0.95) {
            integral_color.w = 1.0;
            break;
        }
    }
    
    //__syncthreads();
    clamp(integral_color,0.0,1.0);
    result[idx*4] = integral_color.x*255;
    result[idx*4+1] = integral_color.y*255;
    result[idx*4+2] = integral_color.z*255;
    result[idx*4+3] = 255;
}

//result will be one of color, JEPG buffer.
extern "C"  
int ray_cast(cudaGLTextureReadOnly& entry_tex, cudaGLTextureReadOnly& exit_tex, int width , int height, 
             cudaVolumeInfos volume_info, cudaRayCastInfos ray_cast_info, unsigned char* d_result, cudaGLTextureWriteOnly& canvas_tex) {
    //1 launch ray cast kernel
    
    CHECK_CUDA_ERROR;
    map_image(entry_tex);
    map_image(exit_tex);
    CHECK_CUDA_ERROR;

    #define BLOCK_SIZE 16
    dim3 block(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(width / BLOCK_SIZE, height / BLOCK_SIZE);
    kernel_ray_cast_main<<<grid, block>>>(entry_tex.cuda_tex_obj, exit_tex.cuda_tex_obj, width, height, volume_info, ray_cast_info, d_result);

    //2 JPEG compress(optional)


    //3 Memcpy device result to device GL texture
    hipDeviceSynchronize();
    CHECK_CUDA_ERROR;

    unmap_image(entry_tex);
    unmap_image(exit_tex);
    CHECK_CUDA_ERROR; 

    map_image(canvas_tex);

    write_image(canvas_tex, d_result, width*height * 4);

    unmap_image(canvas_tex);

    CHECK_CUDA_ERROR;

    return 0;
}

extern "C"
int init_data(cudaVolumeInfos& cuda_volume_infos, unsigned short* data, unsigned int* dim) {
    const unsigned int size = dim[0]*dim[1]*dim[2]*sizeof(unsigned short);

    cuda_volume_infos.dim.x = dim[0];
    cuda_volume_infos.dim.y = dim[1];
    cuda_volume_infos.dim.z = dim[2];

    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(16,0,0,0,hipChannelFormatKindUnsigned);
    CHECK_CUDA_ERROR;

    hipExtent extent;
    extent.width = dim[0];
    extent.height = dim[1];
    extent.depth = dim[2];
    hipMalloc3DArray(&cuda_volume_infos.d_volume_array, &channel_desc, extent);

    CHECK_CUDA_ERROR;

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr = make_hipPitchedPtr((void *)data,extent.width*sizeof(unsigned short), extent.width, extent.height);
    copyParams.dstArray = cuda_volume_infos.d_volume_array;
    copyParams.extent   = extent;
    copyParams.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);

    CHECK_CUDA_ERROR;

    //Cuda resource
    struct hipResourceDesc  res_desc;
    memset(&res_desc, 0, sizeof(hipResourceDesc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = cuda_volume_infos.d_volume_array;
    
    //Texture parameter (like GL's glTexParameteri)
    struct hipTextureDesc tex_desc;
    memset(&tex_desc,0, sizeof(hipTextureDesc));
    tex_desc.addressMode[0] = hipAddressModeClamp;
    tex_desc.addressMode[1] = hipAddressModeClamp;
    tex_desc.addressMode[2] = hipAddressModeClamp;
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = hipReadModeNormalizedFloat;
    tex_desc.normalizedCoords = 1;

    //create texture
    hipTextureObject_t tex_obj = 0;
    hipCreateTextureObject(&cuda_volume_infos.volume_tex_obj, &res_desc, &tex_desc, NULL);

    CHECK_CUDA_ERROR;

    return 0;
}


extern "C"
int init_wl_nonmask(cudaRayCastInfos& ray_cast_infos, float* wl_array_norm) {
    CHECK_CUDA_ERROR;
    hipMalloc(&ray_cast_infos.d_wl_array, sizeof(float)*2);
    hipMemcpy(ray_cast_infos.d_wl_array, wl_array_norm, sizeof(float)*2, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR;
    return 0;
}

extern "C"
int init_lut_nonmask(cudaRayCastInfos& ray_cast_infos, unsigned char* lut_array, int lut_length) {
    //CUDA array
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(
        8, 8, 8, 8, hipChannelFormatKindUnsigned);
    hipMallocArray(&ray_cast_infos.d_lut_array, &channel_desc, lut_length, 1);

    CHECK_CUDA_ERROR;

    //copy data to CUDA array
    hipMemcpyToArray(ray_cast_infos.d_lut_array, 0, 0, lut_array, lut_length * 4, hipMemcpyHostToDevice);

    CHECK_CUDA_ERROR;

    //Cuda resource
    struct hipResourceDesc  res_desc;
    memset(&res_desc, 0, sizeof(hipResourceDesc));
    res_desc.resType = hipResourceTypeArray;
    res_desc.res.array.array = ray_cast_infos.d_lut_array;
    
    //Texture parameter (like GL's glTexParameteri)
    struct hipTextureDesc tex_desc;
    memset(&tex_desc,0, sizeof(hipTextureDesc));
    tex_desc.addressMode[0] = hipAddressModeClamp;
    tex_desc.filterMode = hipFilterModeLinear;
    tex_desc.readMode = hipReadModeNormalizedFloat;
    tex_desc.normalizedCoords = 1;

    //create texture
    hipCreateTextureObject(&ray_cast_infos.lut_tex_obj, &res_desc, &tex_desc, NULL);

    CHECK_CUDA_ERROR; 

    ray_cast_infos.lut_length = lut_length;


    return 0;
}

extern "C"
int init_material_nonmask(cudaRayCastInfos& ray_cast_infos, float* material_array) {
    hipMalloc(&ray_cast_infos.d_material_array, 4*sizeof(float)*3);
    CHECK_CUDA_ERROR; 
    hipMemcpy(ray_cast_infos.d_material_array, material_array, 4*sizeof(float)*3, hipMemcpyHostToDevice);
    CHECK_CUDA_ERROR; 
    return 0;
}